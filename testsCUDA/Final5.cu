#include "hip/hip_runtime.h"
//Library Definition
#include <iostream>  //cout
#include <fstream>   //Files
#include <cstdlib>   //atoi function
//Constant Definition
#define PI 3.141592654
#define blocksize 32

#define n 512
#define p 128

//Print matrix into standard output
void print(double * M,int cols,int rows);
void dot(double * a,double * b, double & c, int cols);
/*
DEVICE FUNCTIONS
*/	
//Matrix transposition (Rows and Cols of M)
__global__ void matrixTrans(double * M,double * MT, int rows, int cols);
//Matrix multiplication(Cols and Rows of the result)
__global__ void matrixMul(double * a,double * b, double * C, int cols,int rows,int cols2);
//INVERSION OF MATRICES ----GAUSS JORDAN METHOD --------
void Inverse(double * A, double * I,int nn);
__global__ void nodiag_normalize(double *A, double *I, int nn, int i);
__global__ void diag_normalize(double *A, double *I, int nn, int i);
__global__ void gaussjordan(double *A, double *I, int nn, int i);
__global__ void set_zero(double *A, double *I, int nn, int i);
//Sum of Matrices
__global__ void matrixSum(const double * M1,const double * M2,double * Msum,double alpha,double beta, int rows, int cols);


//Initialization of matrices, ones, zeros, identity
void set_ones(double * M, int l);
void set_zeros(double * M, int l);
void set_iden(double * M, int l);
//Print matrices into external files
void print_file(char const * NameArch, const double * M,int cols,int rows);
//Random numbers
double normal_rand(void);
/*
MAIN FUNCTION
*/
int main(int argc, char * argv[]){

srand(atoi(argv[1])); //Seed recieved from terminal

//int cols=p;
//int raws=n;
double *X, *Xt, *XXt, *Inv;
double *H0,*H, *J, *Suma;
double *Y,*Yt, *aux, *Id;

int size0 = n * sizeof(double);
int size2 = p * p * sizeof(double);
int size3 = n * n * sizeof(double);
int size4 = n * p * sizeof(double);

hipMallocManaged(&X,size4);
hipMallocManaged(&Xt,size4);
hipMallocManaged(&H0,size4);
hipMallocManaged(&H,size3);
hipMallocManaged(&J,size3);
hipMallocManaged(&Suma,size3);
hipMallocManaged(&XXt,size2);
hipMallocManaged(&Yt,size0);
hipMallocManaged(&Inv,size2);
hipMallocManaged(&Y,size0);
hipMallocManaged(&aux,size0);
hipMallocManaged(&Id,size3);

double ssr=0,sst=0;
double R2=0;
double F=0, Ftest=1.1962078803512777;
for(int row=0;row<n;row++){
        for(int col=0;col<p;col++){
                X[row*p+col]=0.1*normal_rand();//distribution(generator);
                Y[col]=0.1*normal_rand();
        }
}
print_file("x.dat",X,p,n);
print_file("y.dat",Y,1,n);
dim3 threadsPerBlock(blocksize, blocksize);
dim3 numBlocks((n + blocksize - 1) / blocksize, (p + blocksize - 1) / blocksize);
dim3 numBlocks1((p + blocksize - 1) / blocksize, (n + blocksize - 1) / blocksize);
dim3 numBlocks2((n + blocksize - 1) / blocksize, (n + blocksize - 1) / blocksize);
dim3 numBlocks3((1 + blocksize - 1) / blocksize, (n + blocksize - 1) / blocksize);

matrixTrans<<<numBlocks,threadsPerBlock>>>(X,Xt,n,p);
hipDeviceSynchronize();

matrixMul<<<numBlocks1,threadsPerBlock>>>(Xt,X,XXt,p,p,n);
hipDeviceSynchronize();
set_iden(Inv,p);

Inverse(XXt,Inv,p);hipDeviceSynchronize();
//std::cout<<"inv"<<std::endl;
print_file("Inv.dat",Inv,p,p);

//matrixMul<<<numBlocks,threadsPerBlock>>>(X,Xt,XXt,p,p,n);
//hipDeviceSynchronize();

matrixMul<<<numBlocks1,threadsPerBlock>>>(Inv,Xt,H0,n,p,p);
hipDeviceSynchronize();
print_file("H0.dat",H0,n,p);

matrixMul<<<numBlocks2,threadsPerBlock>>>(X,H0,H,n,n,p);
hipDeviceSynchronize();
print_file("H.dat",H,n,n);
set_ones(J,n);
matrixSum<<<numBlocks2,threadsPerBlock>>>(H,J,Suma,1.,-1./n, n,n);
hipDeviceSynchronize();
print_file("Suma.dat",Suma,n,n);

matrixMul<<<1,n>>>(Suma,Y,aux,1,n,p);
hipDeviceSynchronize();
print_file("Aux.dat",aux,n,n);

//matrixMul<<<numBlocks,threadsPerBlock>>>(Y,aux,J,1,1,n);
//hipDeviceSynchronize();
//ssr=J[0];
dot(Y,aux,ssr,n);
set_iden(Id,n);
set_zeros(Suma,n*n);
matrixSum<<<numBlocks,threadsPerBlock>>>(Id,J,Suma,1.,-1./n, n, n);
hipDeviceSynchronize();
set_zeros(aux,n);

matrixMul<<<numBlocks,threadsPerBlock>>>(Suma,Y,aux,n,1,n);
hipDeviceSynchronize();

//matrixMul<<<numBlocks,threadsPerBlock>>>(Y,aux,J,1,1,n);
//hipDeviceSynchronize();
//sst=J[0];
dot(Y,aux,sst,n);

R2=ssr/sst;
F=(R2*(n-p-1.))/((1.-R2)*p);

std::cout<<R2<<' '<<ssr<<' '<<sst<<' '<<F<<std::endl;


hipFree(X);
hipFree(Xt);
hipFree(XXt);
hipFree(Inv);
hipFree(H0);
hipFree(H);
hipFree(J);
hipFree(Suma);
return 0;
}
void print(double * M,int cols,int rows){
  for( int row = 0; row < rows; ++row ){
    for( int col = 0; col < cols; ++col )
      {
	std::cout<<M[col + row*cols]<<'\t';
      }
    std::cout<<"\n";
  }
}
__global__ void matrixTrans(double * M,double * MT, int rows, int cols)
{
  double val=0;
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.y;
  
  if (row < rows && col < cols){   
    val = M[col + row*cols];
    MT[row + col*rows] = val;      
  } 
}
__global__ void matrixMul(double * a,double * b, double * C, int cols,int rows,int cols2)
{
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.y;  
  if (row < rows && col < cols){
C[row*cols+col]  =0;
    for (int k = 0; k < cols2; k++){
	C[row*cols+col]+=b[k*cols+col]*a[row*cols2+k];
   }
  }
}

__global__ void nodiag_normalize(double *A, double *I, int nn, int i){
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if ( x< nn && y < nn){
    if (x < nn && y < nn){
      if (x == i && x!=y){
	I[x*nn + y] /= A[i*nn + i];
	A[x*nn + y] /= A[i*nn + i];
      }
    }
  }	
}

__global__ void diag_normalize(double *A, double *I, int nn, int i){
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;  
  if (x < nn && y < nn){
    if (x == y && x == i){
      I[x*nn + y] /= A[i*nn + i];
      A[x*nn + y] /= A[i*nn + i];
    }
  }
}

__global__ void gaussjordan(double *A, double *I, int nn, int i)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if ( x< nn && y < nn){
    if (x < nn && y < nn){
      if (x != i){
	I[x*nn + y] -= I[i*nn + y] * A[x*nn + i];
	if (y != i){
	  A[x*nn + y] -= A[i*nn + y] * A[x*nn + i];
	}	 
      }
    } 
  }
}

__global__ void set_zero(double *A, double *I, int nn, int i){
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x < nn && y < nn){
    if (x != i){
      if (y == i){
	A[x*nn + y] = 0;
      }
    }
  }
}

void Inverse(double * A, double * I,int nn){
  dim3 threadsPerBlock2(blocksize, blocksize);
  dim3 numBlocks2((nn + blocksize - 1) / blocksize, (nn + blocksize - 1) / blocksize);
  for (int i = 0; i<nn; i++){
    nodiag_normalize << <numBlocks2, threadsPerBlock2 >> >(A, I, nn, i);
    diag_normalize << <numBlocks2, threadsPerBlock2 >> >(A, I, nn, i);
    gaussjordan << <numBlocks2, threadsPerBlock2 >> >(A, I, nn, i);
    set_zero << <numBlocks2, threadsPerBlock2 >> >(A, I, nn, i);
  }
hipDeviceSynchronize();
}

__global__ void matrixSum(const double * M1,const double * M2,double * Msum,double alpha,double beta, int rows, int cols)
{
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.y;  
  if (row < rows && col < cols){   
    Msum[row + col*rows] = alpha*M1[row+col*rows]+beta*M2[row+col*rows];      
  } 
}

void print_file(char const * NameArch, const double * M,int cols,int rows){
  std::ofstream File(NameArch);
File.precision(16);
  for( int row = 0; row < rows; ++row ){
    for( int col = 0; col < cols; ++col )
      {
       	File<<M[col + row*cols]<<'\t';
      }
    File<<"\n";
  }
File.close();
}


// Random number generator as per Abramowitz & Stegun
// Source taken from:
// http://c-faq.com/lib/gaussian.html

double normal_rand(void){
  static double U, V;
  static int phase = 0;
  double Z;
  if(phase == 0) {
    U = (rand() + 1.) / (RAND_MAX + 2.);
    V = rand() / (RAND_MAX + 1.);
    Z = sqrt(-2 * log(U)) * sin(2 * PI * V);
  } else
    Z = sqrt(-2 * log(U)) * cos(2 * PI * V);
  phase = 1 - phase;  
  return Z;
}

void set_iden(double * M, int l){
for(int row=0;row<l;row++){
	for(int col=0;col<l;col++){
	M[row*l+col]=0;
	if (col==row){
		M[row*l+col]=1;
	}
	}
}
}

void set_ones(double * M, int l){
for(int row=0;row<l;row++){
	for(int col=0;col<l;col++){
	M[row*l+col]=1;
	}
}
}
void set_zeros(double * M, int l){
for(int row=0;row<l;row++){
	M[row]=0;
}
}


void dot(double * a,double * b, double & c, int cols){
c=0;
for(int i=0;i<cols;i++){
c+=a[i]*b[i];
}
}

